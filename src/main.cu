#include <chrono> 
#include <time.h>
#include <math.h>
#include "config.h"
#include "stack.cuh"
#include "Sequential.h"
#include "auxFunctions.h"
#include "CSRGraphRep.cuh"
#define USE_GLOBAL_MEMORY 0
#include "LocalStacks.cuh"
#include "GlobalWorkList.cuh"
#include "GlobalWorkListDFS.cuh"

#include "LocalStacksParameterized.cuh"
#include "GlobalWorkListParameterized.cuh"
#undef USE_GLOBAL_MEMORY
#define USE_GLOBAL_MEMORY 1
#include "LocalStacks.cuh"
#include "GlobalWorkList.cuh"
#include "GlobalWorkListDFS.cuh"
#include "LocalStacksParameterized.cuh"
#include "GlobalWorkListParameterized.cuh"
#undef USE_GLOBAL_MEMORY
#include "SequentialParameterized.h"
#include "GlobalWorkListBFS.cuh"

using namespace std;
// BFS
#define MAX_THREADS_PER_GRID (2**31)
#define THREADS_PER_WARP 32
#define THREADS_PER_BLOCK 1024
#define WARPS_PER_BLOCK (THREADS_PER_BLOCK/THREADS_PER_WARP)
#define I_SIZE ((3/2)*THREADS_PER_BLOCK)

int main(int argc, char *argv[]) {

    Config config = parseArgs(argc,argv);
    printf("\nGraph file: %s",config.graphFileName);
    printf("\nUUID: %s\n",config.outputFilePrefix);

    CSRGraph graph = createCSRGraphFromFile(config.graphFileName);
    for (int i = 0; i < graph.vertexNum; ++i)
        printf ("%d %d %d\n", i, graph.degree[i], graph.srcPtr[i]);
    //performChecks(graph, config);
    chrono::time_point<std::chrono::system_clock> begin, end;
	std::chrono::duration<double> elapsed_seconds_max, elapsed_seconds_edge, elapsed_seconds_mvc;
    unsigned int RemoveMaxMinimum = 0;
    unsigned int RemoveEdgeMinimum = 0;

    /*
    begin = std::chrono::system_clock::now(); 
    unsigned int RemoveMaxMinimum = RemoveMaxApproximateMVC(graph);
    end = std::chrono::system_clock::now(); 
	elapsed_seconds_max = end - begin; 

    printf("\nElapsed Time for Approximate Remove Max: %f\n",elapsed_seconds_max.count());
    printf("Approximate Remove Max Minimum is: %u\n", RemoveMaxMinimum);
    fflush(stdout);

    begin = std::chrono::system_clock::now();
    unsigned int RemoveEdgeMinimum = RemoveEdgeApproximateMVC(graph);
    end = std::chrono::system_clock::now(); 
	elapsed_seconds_edge = end - begin; 

    printf("Elapsed Time for Approximate Remove Edge: %f\n",elapsed_seconds_edge.count());
    printf("Approximate Remove Edge Minimum is: %u\n", RemoveEdgeMinimum);
    fflush(stdout);

    unsigned int minimum = (RemoveMaxMinimum < RemoveEdgeMinimum) ? RemoveMaxMinimum : RemoveEdgeMinimum;
    */
    unsigned int minimum = graph.vertexNum;
    unsigned int k = config.k; 
    unsigned int kFound = 0;
    int result = 0;

    if(config.version == SEQUENTIAL){
        /*
        if(config.instance == PVC){
            begin = std::chrono::system_clock::now();
            minimum = SequentialParameterized(graph, minimum, k, &kFound);
            end = std::chrono::system_clock::now(); 
            elapsed_seconds_mvc = end - begin; 
        } else {
            begin = std::chrono::system_clock::now();
            minimum = Sequential(graph, minimum);
            end = std::chrono::system_clock::now(); 
            elapsed_seconds_mvc = end - begin; 
        } 

        printResults(config, RemoveMaxMinimum, RemoveEdgeMinimum, elapsed_seconds_max.count(), elapsed_seconds_edge.count(), minimum, 
            elapsed_seconds_mvc.count(), graph.vertexNum, graph.edgeNum, kFound);

        printf("\nElapsed time: %fs",elapsed_seconds_mvc.count());
        */
    } else {
        hipDeviceSynchronize();

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("\nDevice name: %s\n\n", prop.name);

        int numOfMultiProcessors;
        hipDeviceGetAttribute(&numOfMultiProcessors,hipDeviceAttributeMultiprocessorCount,0);
        printf("NumOfMultiProcessors : %d\n",numOfMultiProcessors);

        int maxThreadsPerMultiProcessor;
        hipDeviceGetAttribute(&maxThreadsPerMultiProcessor,hipDeviceAttributeMaxThreadsPerMultiProcessor,0);
        printf("MaxThreadsPerMultiProcessor : %d\n",maxThreadsPerMultiProcessor);

        int maxThreadsPerBlock;
        hipDeviceGetAttribute(&maxThreadsPerBlock,hipDeviceAttributeMaxThreadsPerBlock,0);
        printf("MaxThreadsPerBlock : %d\n",maxThreadsPerBlock);

        int maxSharedMemPerMultiProcessor;
        hipDeviceGetAttribute(&maxSharedMemPerMultiProcessor,hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,0);
        printf("MaxSharedMemPerMultiProcessor : %d\n",maxSharedMemPerMultiProcessor);

        //setBlockDimAndUseGlobalMemory(config,graph,maxSharedMemPerMultiProcessor,prop.totalGlobalMem, maxThreadsPerMultiProcessor, maxThreadsPerBlock, 
        //    maxThreadsPerMultiProcessor, numOfMultiProcessors, minimum);

        setBlockDimAndUseGlobalMemoryDFS_NoStack(config,graph,maxSharedMemPerMultiProcessor,prop.totalGlobalMem, maxThreadsPerMultiProcessor, maxThreadsPerBlock, 
            maxThreadsPerMultiProcessor, numOfMultiProcessors, graph.vertexNum);
        //performChecks(graph, config);

        printf("\nOur Config :\n");
        int numThreadsPerBlock = config.blockDim;
        int numBlocksPerSm; 
        if (config.useGlobalMemory){
            if (config.version == HYBRID && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkListParameterized_global_kernel, numThreadsPerBlock, 0);
            } else if(config.version == HYBRID && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkList_global_DFS_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacksParameterized_global_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacks_global_kernel, numThreadsPerBlock, 0);
            }
        } else {
            if (config.version == HYBRID && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkListParameterized_shared_kernel, numThreadsPerBlock, 0);
            } else if(config.version == HYBRID && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkList_shared_DFS_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacksParameterized_shared_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacks_shared_kernel, numThreadsPerBlock, 0);
            }
        }

        unsigned int tempNumBlocks;
        if(config.numBlocks){
            tempNumBlocks = config.numBlocks;
        } else {
            tempNumBlocks = numBlocksPerSm*numOfMultiProcessors;
        }

        const unsigned int numBlocks = tempNumBlocks;
        int numThreadsPerSM = numBlocksPerSm * numThreadsPerBlock;
        printf("NumOfThreadPerBlocks : %d\n",numThreadsPerBlock);
        printf("NumOfBlocks : %u\n",numBlocks);
        printf("NumOfBlockPerSM : %d\n",numBlocksPerSm);
        printf("NumOfThreadsPerSM : %d\n\n",numThreadsPerSM);
        fflush(stdout);
        graph.numBlocks = numBlocks;
        //Allocate NODES_PER_SM
        int * NODES_PER_SM_d;
        #if USE_COUNTERS
            int * NODES_PER_SM;
            NODES_PER_SM = (int *)malloc(sizeof(int)*numOfMultiProcessors);
            for (unsigned int i = 0;i<numOfMultiProcessors;++i){
                NODES_PER_SM[i]=0;
            }
            hipMalloc((void**)&NODES_PER_SM_d, numOfMultiProcessors*sizeof(int));
            hipMemcpy(NODES_PER_SM_d, NODES_PER_SM, numOfMultiProcessors*sizeof(int), hipMemcpyHostToDevice);
        #endif

        // Allocate GPU graph
        CSRGraph graph_d = allocateGraph(graph);
        DFSWorkList dfsWL_d =  allocateDFSWorkList(graph);
        printf("Allocated graph\n");
        hipError_t erra = hipDeviceSynchronize();
        if(erra != hipSuccess) {
            printf("GPU Error: %s\n", hipGetErrorString(erra));
            exit(1);
        }
        // Allocate GPU stack
        Stacks stacks_d;
        //stacks_d = allocateStacks(graph.vertexNum,numBlocks,minimum);

        //Global Entries Memory Allocation
        int * global_memory_d;
        if(config.useGlobalMemory){
            //hipMalloc((void**)&global_memory_d, sizeof(int)*graph.vertexNum*numBlocks*2);
        }

        unsigned int * minimum_d;
        hipMalloc((void**) &minimum_d, sizeof(unsigned int));

        // Allocate counter for each block
        Counters* counters_d;
        hipMalloc((void**)&counters_d, numBlocks*sizeof(Counters));

        // Copy minimum
        hipMemcpy(minimum_d, &minimum, sizeof(unsigned int), hipMemcpyHostToDevice);

        unsigned int *k_d = NULL;
        unsigned int *kFound_d = NULL;
        if(config.instance == PVC){
            hipMalloc((void**)&k_d, sizeof(unsigned int));
            hipMemcpy(k_d, &k, sizeof(unsigned int), hipMemcpyHostToDevice);

            hipMalloc((void**)&kFound_d, sizeof(unsigned int));
            hipMemcpy(kFound_d, &kFound, sizeof(unsigned int), hipMemcpyHostToDevice);
        }

        // HYBRID
        // Allocate GPU queue
        WorkList workList_d;
        //First to dequeue flag
        int *first_to_dequeue_global_d;
        int first_to_dequeue_global=0;
        // STACKONLY
        unsigned int * pathCounter_d;
        unsigned int pathCounter = 0;
        if(config.version == HYBRID){
            hipMalloc((void**)&first_to_dequeue_global_d, sizeof(int));
            hipMemcpy(first_to_dequeue_global_d, &first_to_dequeue_global, sizeof(int), hipMemcpyHostToDevice);
            //workList_d =  allocateWorkList(graph, config, numBlocks);    
        } else {
            hipMalloc((void**)&pathCounter_d, sizeof(unsigned int));
            hipMemcpy(pathCounter_d, &pathCounter, sizeof(unsigned int), hipMemcpyHostToDevice);
        }

        int sharedMemNeeded = graph.vertexNum;
        if(graph.vertexNum > numThreadsPerBlock*2){
            sharedMemNeeded+=graph.vertexNum;
        } else {
            sharedMemNeeded+=numThreadsPerBlock*2;
        }
        sharedMemNeeded *= sizeof(int);
        
        hipEvent_t start, stop;
        hipDeviceSynchronize();
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);


        #if USE_GLOBAL_MEMORY
        GlobalDFSKernelArgs args;
        args.global_memory = global_memory_d; 
        #else
        SharedDFSKernelArgs args;
        #endif
        args.stacks = stacks_d; 
        args.minimum = minimum_d; 
        args.workList = workList_d; 
        args.dfsWL = dfsWL_d;
        args.graph = graph_d; 
        args.counters = counters_d; 
        args.first_to_dequeue_global = first_to_dequeue_global_d; 
        args.NODES_PER_SM = NODES_PER_SM_d;
        void *kernel_args[] = {&args};
        printf("Launching kernel\n");
        int dimGridBFS = (graph.vertexNum + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
        if (config.useGlobalMemory){
            if (config.version == HYBRID && config.instance==PVC){
                GlobalWorkListParameterized_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, global_memory_d, k_d, kFound_d, NODES_PER_SM_d);
            } else if(config.version == HYBRID && config.instance==MVC) {
                bool pathFound = false;
                bool pathFoundOnAnyIteration = false;
                int iter = 0;
                do {
                    printf("Iter %d\n",iter++);
                    hipMemset(&graph_d.foundPath[0], false, sizeof(bool));
                    pathFound = false;
                    pathFoundOnAnyIteration = false;
                    unsigned int depth = 0;
                    GlobalWorkList_Set_Sources_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d);
                    for (; depth < graph.vertexNum && !pathFound; ++depth){ 
                        GlobalWorkList_BFS_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d, depth);
                        GlobalWorkList_Extract_Bridges_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, dfsWL_d,  graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d, depth);
                        hipLaunchCooperativeKernel((void*)(GlobalWorkList_global_DFS_kernel), numBlocks, numThreadsPerBlock, kernel_args) ;
                        hipDeviceSynchronize();
                        hipMemcpy(&pathFound, &graph_d.foundPath[0], sizeof(bool), hipMemcpyDeviceToHost);
                        //GlobalWorkList_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, global_memory_d, NODES_PER_SM_d);
                        printf("Found path %s\n",pathFound?"True":"False");
                        pathFoundOnAnyIteration |= pathFound;
                    }
                    graph_d.reset();
                    dfsWL_d.reset();
                    //exit(1);
                } while(pathFoundOnAnyIteration);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                LocalStacksParameterized_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, graph_d, global_memory_d, k_d, kFound_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                LocalStacks_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, graph_d, minimum_d, global_memory_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            }
        } else {
            if (config.version == HYBRID && config.instance==PVC){
                GlobalWorkListParameterized_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, k_d, kFound_d, NODES_PER_SM_d);
            } else if(config.version == HYBRID && config.instance==MVC) {
                bool pathFound = false;
                bool pathFoundOnAnyIteration = false;
                int iter = 0;
                do {
                    printf("Iter %d\n",iter++);
                    hipMemset(&graph_d.foundPath[0], false, sizeof(bool));
                    pathFound = false;
                    pathFoundOnAnyIteration = false;
                    unsigned int depth = 0;
                    GlobalWorkList_Set_Sources_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d);
                    for (; depth < graph.vertexNum && !pathFound; ++depth){ 
                        GlobalWorkList_BFS_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d, depth);
                        GlobalWorkList_Extract_Bridges_kernel <<< dimGridBFS, THREADS_PER_BLOCK >>> (stacks_d, minimum_d, workList_d, dfsWL_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d, depth);
                        hipLaunchCooperativeKernel((void*)(GlobalWorkList_shared_DFS_kernel), numBlocks, numThreadsPerBlock, kernel_args) ;
                        hipDeviceSynchronize();
                        hipMemcpy(&pathFound, &graph_d.foundPath[0], sizeof(bool), hipMemcpyDeviceToHost);
                        //GlobalWorkList_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, global_memory_d, NODES_PER_SM_d);
                        printf("Found path %s\n",pathFound?"True":"False");
                        pathFoundOnAnyIteration |= pathFound;
                    }
                    graph_d.reset();
                    dfsWL_d.reset();
                    //exit(1);

                } while(pathFoundOnAnyIteration);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                LocalStacksParameterized_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, graph_d, k_d, kFound_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                LocalStacks_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, graph_d, minimum_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            }
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipDeviceSynchronize();
        if(err != hipSuccess) {
            printf("GPU Error: %s\n", hipGetErrorString(err));
            exit(1);
        }

        // Copy back result
        if(config.instance == PVC){
            hipMemcpy(&kFound, kFound_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        } else {
            hipMemcpy(&minimum, minimum_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Elapsed time: %fms \n", milliseconds);

        printResults(config, RemoveMaxMinimum, RemoveEdgeMinimum, elapsed_seconds_max.count(), elapsed_seconds_edge.count(), minimum, milliseconds, numBlocks, 
            numBlocksPerSm, numThreadsPerSM, graph.vertexNum-1, graph.edgeNum, kFound);
        thrust::device_ptr<int> m_vec=thrust::device_pointer_cast(graph_d.matching);
        using namespace thrust::placeholders;
        result = thrust::count_if(m_vec, m_vec+graph.vertexNum, _1 > -1);
        #if USE_COUNTERS
        printCountersInFile(config,counters_d,numBlocks);
        printNodesPerSM(config,NODES_PER_SM_d,numOfMultiProcessors);
        hipFree(NODES_PER_SM);
        #endif

        if(config.instance == PVC){
            hipFree(k_d);
        }
        graph.del();
        hipFree(minimum_d);
        hipFree(counters_d);
        cudaFreeGraph(graph_d);

        cudaFreeStacks(stacks_d);
        
        #if USE_GLOBAL_MEMORY
        hipFree(global_memory_d);
        #endif

        if(config.version == HYBRID){
            //cudaFreeWorkList(workList_d);
            hipFree(first_to_dequeue_global_d);
        } else {
            hipFree(pathCounter_d);
        }

    }

    if(config.instance == PVC){
        if(kFound){
            printf("\nMinimum is less than or equal to K: %u\n\n",k);
        } else {
            printf("\nMinimum is greater than K: %u\n\n",k);
        }
    } else {
        printf("\nSize of matching: %u\n\n", result/2);
    }

    return 0;
}