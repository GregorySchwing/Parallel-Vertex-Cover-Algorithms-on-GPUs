/*
 * This source code is distributed under the terms defined  
 * in the file bfstdcsc_main.c of this source distribution.
 */
/* 
 *  Breadth first search (BFS) 
 *  Single precision (float data type) 
 *  TurboBFS_CSC_TD:bfsgputdcsc_sc.cu
 * 
 *  This program computes the GPU-based parallel 
 *  top-down BFS (scalar) for unweighted graphs represented 
 *  by sparse adjacency matrices in the CSC format, including
 *  the computation of the S array to store the depth at 
 *  which each vertex is discovered.  
 *
 */

#include <cstdlib>
#include <iostream>
#include <cassert>
#include <cmath>

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "bfstdcsc.h"

Match_GPU create_match_gpu_struct(int N){

  Match_GPU match_device;

  /*Allocate device memory for the vector req_d */
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&match_device.req_d),sizeof(*match_device.req_d)*(N)));

  /*allocate unified memory for integer variable c for control of while loop*/
  checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&match_device.c),sizeof(*match_device.c)));

  return match_device;
}