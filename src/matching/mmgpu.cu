/*
 * This source code is distributed under the terms defined  
 * in the file bfstdcsc_main.c of this source distribution.
 */
/* 
 *  Breadth first search (BFS) 
 *  Single precision (float data type) 
 *  TurboBFS_CSC_TD:bfsgputdcsc_sc.cu
 * 
 *  This program computes the GPU-based parallel 
 *  top-down BFS (scalar) for unweighted graphs represented 
 *  by sparse adjacency matrices in the CSC format, including
 *  the computation of the S array to store the depth at 
 *  which each vertex is discovered.  
 *
 */

#include <cstdlib>
#include <iostream>
#include <cassert>
#include <cmath>

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "matchgpu.h"
#include "mis_kernels.h"

#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

extern "C"{
                 #include "bfstdcsc.h"

}

struct is_less_than_0
{
  __host__ __device__
  int operator()(int &x)
  {
    if (x > -1)
      return x;
    else
      return -1;
  }
};

/* 
 * Function to compute a gpu-based parallel maximal matching for 
 * unweighted graphs represented by sparse adjacency matrices in CSC format.
 *  
 */
//int  mm_gpu_csc (int *IC_h,int *CP_h,int *m_h,int *_m_d,int *req_h,int *c_h,int nz,int n,int repetition, int exec_protocol){
int mm_gpu_csc(struct Graph * graph,struct match * m, int exec_protocol){
//int  mm_gpu_csc (int *IC_h,int *CP_h,int *m_h,int *_m_d,int *req_h,int *c_h,int nz,int n,int repetition, int exec_protocol){
  float t_mm;
  float t_mm_t = 0.0;
  float t_thrust;
  float t_thrust_t = 0.0;
  int i,dimGrid;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  int *CP_h = graph->CscA.CP;
  int *IC_h = graph->CscA.IC;
  int *m_h = m->m_h;
  int *m_hgpu = m->m_hgpu;
  int n = graph->N;
  int nz = graph->nz;
  int repetition = graph->repet;

  int *CP_d;
  int *IC_d;
  int *m_d;
  int *req_d;
  int *c;
  int result, resultSum = 0;


  if (exec_protocol){
    CP_d = graph->graph_device.CP_d;
    IC_d = graph->graph_device.IC_d;
    m_d = m->match_device.m_d;
    req_d = m->match_device.req_d;
    c = m->match_device.c;
  } else {
    /*Allocate device memory for the vector CP_d */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&CP_d),sizeof(*CP_d)*(n+1)));
    /*Copy host memory (CP_h) to device memory (CP_d)*/
    checkCudaErrors(hipMemcpy(CP_d,CP_h,(n+1)*sizeof(*CP_d),hipMemcpyHostToDevice));

    /*Allocate device memory for the vector IC_d */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&IC_d),sizeof(*IC_d)*nz));
    /*Copy host memory (IC_h) to device memory (IC_d)*/
    checkCudaErrors(hipMemcpy(IC_d,IC_h,nz*sizeof(*IC_d),hipMemcpyHostToDevice));

    /*Allocate device memory for the vector m_d, and set m_d to zero. */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&m_d),sizeof(*m_d)*n));
    checkCudaErrors(hipMemset(m_d,0,sizeof(*m_d)*n));

    /*Allocate device memory for the vector f_d*/
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&req_d),sizeof(*req_d)*n));

    /*allocate unified memory for integer variable c for control of while loop*/
    checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&c),sizeof(*c)));
  }


  srand(1);
  /*computing MM */
  dimGrid = (n + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  for (i = 0; i<repetition; i++){
    *c = 1;
    //d = 0;
    checkCudaErrors(hipMemset(req_d,0,sizeof(*req_d)*n));
    checkCudaErrors(hipMemset(m_d,0,sizeof(*m_d)*n));
    int count = 0;
    while (*c && ++count < NR_MAX_MATCH_ROUNDS){
      //d = d + 1;
      *c = 0;
      hipEventRecord(start);
      gaSelect<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, c, n, rand());
      grRequest<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
      grRespond<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
      gMatch<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, req_d, n);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_mm,start,stop);
      t_mm_t += t_mm;
    }
    hipEventRecord(start);
    using namespace thrust::placeholders;
    thrust::device_ptr<int> m_vec=thrust::device_pointer_cast(m_d);
    thrust::for_each(m_vec, m_vec+n, _1 -= 4);
    
    result = thrust::count(m_vec, m_vec+n, -1);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_thrust,start,stop);
    t_thrust_t += t_thrust;
    resultSum += result;

  }
  printf("\nbfs_gpu_mm_csc_sc::t_sum=%lfms \n",t_mm_t+t_thrust_t);
  
  int print_t = 1;
  if (print_t){
    printf("mm_gpu_csc::average time mm = %lfms, avg matched %f, avg unmatched %f, total %d\n",(t_mm_t+t_thrust_t)/repetition, 
      (float)n-((float)resultSum/(float)repetition), ((float)resultSum/(float)repetition), n);
  }
  /*cleanup memory*/
  if (exec_protocol){
    // Really only necessary if checking against seq malt bfs
    checkCudaErrors(hipMemcpy(m_h,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(m_hgpu,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));

  } else {
    /*Copy device memory (m_d) to host memory (S_h)*/
    checkCudaErrors(hipMemcpy(m_h,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(CP_d));
    checkCudaErrors(hipFree(IC_d));
    checkCudaErrors(hipFree(m_d));
    checkCudaErrors(hipFree(req_d));
    checkCudaErrors(hipFree(c));
  }
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return 0;
}//end bfs_gpu_td_csc_sc

int mm_gpu_csc_from_mis(struct Graph * graph,struct match * m,struct MIS * mis, int exec_protocol){
//int  mm_gpu_csc (int *IC_h,int *CP_h,int *m_h,int *_m_d,int *req_h,int *c_h,int nz,int n,int repetition, int exec_protocol){
  float t_mm;
  float t_mm_t = 0.0;
  float t_thrust;
  float t_thrust_t = 0.0;
  float t_mis;
  float t_mis_t = 0.0;
  int i,dimGrid;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


  int *CP_h = graph->CscA.CP;
  int *IC_h = graph->CscA.IC;
  int *m_h = m->m_h;
  int *m_hgpu = m->m_hgpu;
  int n = graph->N;
  int nz = graph->nz;
  int repetition = graph->repet;

  int *CP_d;
  int *IC_d;
  int *m_d;
  int *req_d;
  int *L_d;
  int *c;
  int result, resultSum = 0;

  if (exec_protocol){
    CP_d = graph->graph_device.CP_d;
    IC_d = graph->graph_device.IC_d;
    m_d = m->match_device.m_d;
    req_d = m->match_device.req_d;
    L_d = mis->mis_device.L_d;
    c = m->match_device.c;
  } else {
    /*Allocate device memory for the vector CP_d */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&CP_d),sizeof(*CP_d)*(n+1)));
    /*Copy host memory (CP_h) to device memory (CP_d)*/
    checkCudaErrors(hipMemcpy(CP_d,CP_h,(n+1)*sizeof(*CP_d),hipMemcpyHostToDevice));

    /*Allocate device memory for the vector IC_d */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&IC_d),sizeof(*IC_d)*nz));
    /*Copy host memory (IC_h) to device memory (IC_d)*/
    checkCudaErrors(hipMemcpy(IC_d,IC_h,nz*sizeof(*IC_d),hipMemcpyHostToDevice));

    /*Allocate device memory for the vector m_d, and set m_d to zero. */
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&m_d),sizeof(*m_d)*n));
    checkCudaErrors(hipMemset(m_d,0,sizeof(*m_d)*n));

    /*Allocate device memory for the vector f_d*/
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&req_d),sizeof(*req_d)*n));

    /*allocate unified memory for integer variable c for control of while loop*/
    checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&c),sizeof(*c)));
  }

  using namespace thrust::placeholders;
  thrust::device_ptr<int> m_vec=thrust::device_pointer_cast(m_d);

  srand(1);
  /*computing MM */
  dimGrid = (n + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  thrust::device_ptr<int> L_vec=thrust::device_pointer_cast(L_d);
  int L_size, L_sum=0;
  for (i = 0; i<repetition; i++){
    *c = 1;
    //d = 0;
    checkCudaErrors(hipMemset(req_d,0,sizeof(*req_d)*n));
    checkCudaErrors(hipMemset(m_d,0,sizeof(*m_d)*n));

    *c = 1;
    int lastMatch =0;
    result = 0;
    do {
      lastMatch = result;
      checkCudaErrors(hipMemset(L_d,0,sizeof(*L_d)*graph->N));
      while(*c){
        *c = 0;
        hipEventRecord(start);
        set_L_unmatched<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d, IC_d, L_d, m_d,c, graph->N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t_mis,start,stop);
        t_mis_t += t_mis;
      }
      L_size = thrust::count(L_vec, L_vec+graph->N, 1);
      //printf("\nMIS count::t_sum=%dms \n",L_size);

      L_sum += L_size;

      int count = 0;
      *c = 1;
      while (*c && ++count < NR_MAX_MATCH_ROUNDS){
        //d = d + 1;
        *c = 0;
        hipEventRecord(start);
        gaSelect_from_mis<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, c, L_d,n);
        grRequest<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
        grRespond<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
        gMatch<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, req_d, n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&t_mm,start,stop);
        t_mm_t += t_mm;
      }
      result = thrust::count_if(m_vec, m_vec+n, _1>3);
      //printf("\nmatch count::t_sum=%dms \n",result);
    }while(result!=lastMatch);
    int count = 0;
    *c = 1;
    while (*c && ++count < NR_MAX_MATCH_ROUNDS){
      //d = d + 1;
      *c = 0;
      hipEventRecord(start);
      gaSelect<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, c,n,rand());
      grRequest<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
      grRespond<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d,IC_d,req_d, m_d, n);
      gMatch<<<dimGrid,THREADS_PER_BLOCK>>>(m_d, req_d, n);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_mm,start,stop);
      t_mm_t += t_mm;
    }
    hipEventRecord(start);
    using namespace thrust::placeholders;
    thrust::device_ptr<int> m_vec=thrust::device_pointer_cast(m_d);
    thrust::for_each(m_vec, m_vec+n, _1 -= 4);
    //thrust::for_each(m_vec, m_vec+n, is_less_than_0());

    result = thrust::count_if(m_vec, m_vec+n, _1<0);
    int badresult = thrust::count_if(m_vec, m_vec+n, _1<-1);
    if (badresult){
      printf ("Bad results %d\n", badresult);
      exit(1);
    }
    //result = thrust::count(m_vec, m_vec+n, -1);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&t_thrust,start,stop);
    t_thrust_t += t_thrust;
    resultSum += result;
    
  }
  printf("\nmm_gpu_csc_from_mis::t_sum=%lfms \n",t_mm_t+t_thrust_t);
  
  int print_t = 1;
  if (print_t){
    printf("mm_gpu_csc_from_mis::average time mm = %lfms, avg matched %f, avg unmatched %f, total %d\n",(t_mm_t+t_thrust_t)/repetition, 
      (float)n-((float)resultSum/(float)repetition), ((float)resultSum/(float)repetition), n);
  }
  /*cleanup memory*/
  if (exec_protocol){
    // Really only necessary if checking against seq malt bfs
    checkCudaErrors(hipMemcpy(m_h,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(m_hgpu,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));

  } else {
    /*Copy device memory (m_d) to host memory (S_h)*/
    checkCudaErrors(hipMemcpy(m_h,m_d, n*sizeof(*m_d),hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(CP_d));
    checkCudaErrors(hipFree(IC_d));
    checkCudaErrors(hipFree(m_d));
    checkCudaErrors(hipFree(req_d));
    checkCudaErrors(hipFree(c));
  }
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return 0;
}//end bfs_gpu_td_csc_sc



/* 
 * Function to compute a gpu-based parallel maximal matching for 
 * unweighted graphs represented by sparse adjacency matrices in CSC format.
 *  
 */
void add_edges_to_unmatched_from_last_vertex_gpu_csc(struct Graph * graph,struct match * m,int exec_protocol){

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Local copies are only used when exec_protocol is 0
  // This is turning into a debug mode execution.
  /*Allocate device memory for the vector CP_d */
  int *CP_d;
  /*Allocate device memory for the vector IC_d */
  int *IC_d;
  /*Allocate device memory for the vector m_d, and set m_d to zero. */
  int *m_d;

  if (exec_protocol){
    CP_d = graph->graph_device.CP_d;
    IC_d = graph->graph_device.IC_d;
    m_d = m->match_device.m_d;
  } else {
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&CP_d),sizeof(*CP_d)*(graph->N_including_supersource+1)));
    /*Copy host memory (CP_h) to device memory (CP_d)*/
    checkCudaErrors(hipMemcpy(CP_d,graph->CscA.CP,(graph->N+1)*sizeof(*CP_d),hipMemcpyHostToDevice));


    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&IC_d),sizeof(*IC_d)*graph->nz_including_supersource));
    /*Copy host memory (IC_h) to device memory (IC_d)*/
    checkCudaErrors(hipMemcpy(IC_d,graph->CscA.IC,graph->nz*sizeof(*IC_d),hipMemcpyHostToDevice));


    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&m_d),sizeof(*m_d)*graph->N));
    checkCudaErrors(hipMemcpy(m_d,m->m_h,graph->N*sizeof(*m_d),hipMemcpyHostToDevice));
  }

  thrust::device_ptr<int> m_vec=thrust::device_pointer_cast(m_d);

  using namespace thrust;
  using namespace thrust::placeholders;

 // storage for the nonzero indices
 thrust::device_vector<int> indices(graph->N);
 // compute indices of nonzero elements
 typedef thrust::device_vector<int>::iterator IndexIterator;
 IndexIterator indices_end = thrust::copy_if(thrust::make_counting_iterator(0),
                                             thrust::make_counting_iterator(graph->N),
                                             m_vec,
                                             indices.begin(),
                                             _1 == -1);

  printf("\nnum unmatched %d\n", indices_end-indices.begin());
  m->num_matched_h[0] = graph->N-(indices_end-indices.begin());
  // Set number of sources
  thrust::device_ptr<int> CP_vec=thrust::device_pointer_cast(CP_d);

  // NumSources = (indices_end-indices.begin())
  // Previous index = nz
  // Prefix sum = previous index + NumSources
  thrust::device_vector<int> numSourcesPrefixSum(1);

  numSourcesPrefixSum[0] = graph->nz+(indices_end-indices.begin());

  // CP is int[N+2], N_including_supersource = N+1, CP[N+1]=num unmatched+nz
  thrust::copy(thrust::device, numSourcesPrefixSum.begin(), numSourcesPrefixSum.end(), CP_vec+graph->N_including_supersource);

  // Set sources
  thrust::device_ptr<int> IC_vec=thrust::device_pointer_cast(IC_d);

  // IC is int[nz+((N+1)/2)], 
  // IC[nz]...IC[nz+num unmatched] = sources
  //printf("Max edge count %d\n", nz_including_supersource);
  //printf("Curr edge count %d\n", nz);
  //printf("New edge count %d\n", nz+(indices_end-indices.begin()));

  // Copy sources into column array IC at CP[N] to CP[N+1]
  thrust::copy(thrust::device, indices.begin(), indices_end, IC_vec+graph->nz);

  if (graph->seq){
    checkCudaErrors(hipMemcpy(graph->CscA_hgpu.CP,CP_d,(graph->N_including_supersource+1)*sizeof(*CP_d),hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(graph->CscA_hgpu.IC,IC_d,graph->nz_including_supersource*sizeof(*IC_d),hipMemcpyDeviceToHost));

  }

  if (exec_protocol){

  } else {
    checkCudaErrors(hipFree(CP_d));
    checkCudaErrors(hipFree(IC_d));
    checkCudaErrors(hipFree(m_d));
  }
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  return;
}//end bfs_gpu_td_csc_sc