#include <chrono> 
#include <time.h>
#include <math.h>
#include <stdexcept>
#include "config.h"
#include "stack.cuh"
#include "Sequential.h"
#include "auxFunctions.h"
#include "CSRGraphRep.cuh"
#define USE_GLOBAL_MEMORY 0
#include "LocalStacks.cuh"
#include "GlobalWorkList.cuh"
#include "LocalStacksParameterized.cuh"
#include "GlobalWorkListParameterized.cuh"
#undef USE_GLOBAL_MEMORY
#define USE_GLOBAL_MEMORY 1
#include "LocalStacks.cuh"
#include "GlobalWorkList.cuh"
#include "LocalStacksParameterized.cuh"
#include "GlobalWorkListParameterized.cuh"
#undef USE_GLOBAL_MEMORY
#include "SequentialParameterized.h"
#include "KernelArgs.cuh"

#include <hip/hip_runtime_api.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_cooperative_groups.h>
using namespace std;



int main(int argc, char *argv[]) {
    KernelArgs fbArgs;

    Config config = parseArgs(argc,argv);
    printf("\nGraph file: %s",config.graphFileName);
    printf("\nUUID: %s\n",config.outputFilePrefix);

    CSRGraph graph = createCSRGraphFromFile(config.graphFileName);
    performChecks(graph, config);

    chrono::time_point<std::chrono::system_clock> begin, end;
	std::chrono::duration<double> elapsed_seconds_max, elapsed_seconds_edge, elapsed_seconds_mvc;

    begin = std::chrono::system_clock::now(); 
    unsigned int RemoveMaxMinimum = RemoveMaxApproximateMVC(graph);
    end = std::chrono::system_clock::now(); 
	elapsed_seconds_max = end - begin; 

    printf("\nElapsed Time for Approximate Remove Max: %f\n",elapsed_seconds_max.count());
    printf("Approximate Remove Max Minimum is: %u\n", RemoveMaxMinimum);
    fflush(stdout);

    begin = std::chrono::system_clock::now();
    unsigned int RemoveEdgeMinimum = RemoveEdgeApproximateMVC(graph);
    end = std::chrono::system_clock::now(); 
	elapsed_seconds_edge = end - begin; 

    printf("Elapsed Time for Approximate Remove Edge: %f\n",elapsed_seconds_edge.count());
    printf("Approximate Remove Edge Minimum is: %u\n", RemoveEdgeMinimum);
    fflush(stdout);

    unsigned int minimum = (RemoveMaxMinimum < RemoveEdgeMinimum) ? RemoveMaxMinimum : RemoveEdgeMinimum;

    unsigned int k = config.k; 
    unsigned int kFound = 0;

    if(config.version == SEQUENTIAL){
        if(config.instance == PVC){
            begin = std::chrono::system_clock::now();
            minimum = SequentialParameterized(graph, minimum, k, &kFound);
            end = std::chrono::system_clock::now(); 
            elapsed_seconds_mvc = end - begin; 
        } else {
            begin = std::chrono::system_clock::now();
            minimum = Sequential(graph, minimum);
            end = std::chrono::system_clock::now(); 
            elapsed_seconds_mvc = end - begin; 
        } 

        printResults(config, RemoveMaxMinimum, RemoveEdgeMinimum, elapsed_seconds_max.count(), elapsed_seconds_edge.count(), minimum, 
            elapsed_seconds_mvc.count(), graph.vertexNum, graph.edgeNum, kFound);

        printf("\nElapsed time: %fs",elapsed_seconds_mvc.count());
    } else {
        hipDeviceSynchronize();

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("\nDevice name: %s\n\n", prop.name);

        int numOfMultiProcessors;
        hipDeviceGetAttribute(&numOfMultiProcessors,hipDeviceAttributeMultiprocessorCount,0);
        printf("NumOfMultiProcessors : %d\n",numOfMultiProcessors);

        int maxThreadsPerMultiProcessor;
        hipDeviceGetAttribute(&maxThreadsPerMultiProcessor,hipDeviceAttributeMaxThreadsPerMultiProcessor,0);
        printf("MaxThreadsPerMultiProcessor : %d\n",maxThreadsPerMultiProcessor);

        int maxThreadsPerBlock;
        hipDeviceGetAttribute(&maxThreadsPerBlock,hipDeviceAttributeMaxThreadsPerBlock,0);
        printf("MaxThreadsPerBlock : %d\n",maxThreadsPerBlock);

        int maxSharedMemPerMultiProcessor;
        hipDeviceGetAttribute(&maxSharedMemPerMultiProcessor,hipDeviceAttributeMaxSharedMemoryPerMultiprocessor,0);
        printf("MaxSharedMemPerMultiProcessor : %d\n",maxSharedMemPerMultiProcessor);

        setBlockDimAndUseGlobalMemory(config,graph,maxSharedMemPerMultiProcessor,prop.totalGlobalMem, maxThreadsPerMultiProcessor, maxThreadsPerBlock, 
            maxThreadsPerMultiProcessor, numOfMultiProcessors, minimum);
        performChecks(graph, config);

        printf("\nOur Config :\n");
        int numThreadsPerBlock = config.blockDim;
        int numBlocksPerSm; 

        int sharedMemNeeded = graph.vertexNum;
        if(graph.vertexNum > numThreadsPerBlock*2){
            sharedMemNeeded+=graph.vertexNum;
        } else {
            sharedMemNeeded+=numThreadsPerBlock*2;
        }
        sharedMemNeeded *= sizeof(int);

        if (config.useGlobalMemory){
            if (config.version == HYBRID && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkListParameterized_global_kernel, numThreadsPerBlock, 0);
            } else if(config.version == HYBRID && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkList_global_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacksParameterized_global_kernel, numThreadsPerBlock, 0);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacks_global_kernel, numThreadsPerBlock, 0);
            }
        } else {
            if (config.version == HYBRID && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkListParameterized_shared_kernel, numThreadsPerBlock, sharedMemNeeded);
            } else if(config.version == HYBRID && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, GlobalWorkList_shared_kernel, numThreadsPerBlock, sharedMemNeeded);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacksParameterized_shared_kernel, numThreadsPerBlock, sharedMemNeeded);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, LocalStacks_shared_kernel, numThreadsPerBlock, sharedMemNeeded);
            }
        }

        unsigned int tempNumBlocks;
        if(config.numBlocks){
            tempNumBlocks = config.numBlocks;
        } else {
            tempNumBlocks = numBlocksPerSm*numOfMultiProcessors;
        }

        const unsigned int numBlocks = tempNumBlocks;
        int numThreadsPerSM = numBlocksPerSm * numThreadsPerBlock;
        printf("NumOfThreadPerBlocks : %d\n",numThreadsPerBlock);
        printf("NumOfBlocks : %u\n",numBlocks);
        printf("NumOfBlockPerSM : %d\n",numBlocksPerSm);
        printf("NumOfThreadsPerSM : %d\n\n",numThreadsPerSM);
        fflush(stdout);

        //Allocate NODES_PER_SM
        int * NODES_PER_SM_d;
        #if USE_COUNTERS
            int * NODES_PER_SM;
            NODES_PER_SM = (int *)malloc(sizeof(int)*numOfMultiProcessors);
            for (unsigned int i = 0;i<numOfMultiProcessors;++i){
                NODES_PER_SM[i]=0;
            }
            hipMalloc((void**)&NODES_PER_SM_d, numOfMultiProcessors*sizeof(int));
            hipMemcpy(NODES_PER_SM_d, NODES_PER_SM, numOfMultiProcessors*sizeof(int), hipMemcpyHostToDevice);
        #endif

        // Allocate GPU graph
        CSRGraph graph_d;
        fbArgs.graph = allocateGraph(graph);

        // Allocate GPU stack
        Stacks stacks_d;
        fbArgs.stacks = allocateStacks(graph.vertexNum,numBlocks,minimum);

        //Global Entries Memory Allocation
        int * global_memory_d;
        if(config.useGlobalMemory){
            hipMalloc((void**)&global_memory_d, sizeof(int)*graph.vertexNum*numBlocks*2);
        }

        unsigned int * minimum_d;
        hipMalloc((void**) &minimum_d, sizeof(unsigned int));

        // Allocate counter for each block
        Counters* counters_d;
        hipMalloc((void**)&counters_d, numBlocks*sizeof(Counters));

        // Copy minimum
        hipMemcpy(minimum_d, &minimum, sizeof(unsigned int), hipMemcpyHostToDevice);

        unsigned int *k_d = NULL;
        unsigned int *kFound_d = NULL;
        if(config.instance == PVC){
            hipMalloc((void**)&k_d, sizeof(unsigned int));
            hipMemcpy(k_d, &k, sizeof(unsigned int), hipMemcpyHostToDevice);

            hipMalloc((void**)&kFound_d, sizeof(unsigned int));
            hipMemcpy(kFound_d, &kFound, sizeof(unsigned int), hipMemcpyHostToDevice);
        }

        // HYBRID
        // Allocate GPU queue
        WorkList workList_d;
        //First to dequeue flag
        int *first_to_dequeue_global_d;
        int first_to_dequeue_global=0;
        // STACKONLY
        unsigned int * pathCounter_d;
        unsigned int pathCounter = 0;
        if(config.version == HYBRID){
            hipMalloc((void**)&first_to_dequeue_global_d, sizeof(int));
            hipMemcpy(first_to_dequeue_global_d, &first_to_dequeue_global, sizeof(int), hipMemcpyHostToDevice);
            fbArgs.workList =  allocateWorkList(graph, config, numBlocks);    
        } else {
            hipMalloc((void**)&pathCounter_d, sizeof(unsigned int));
            hipMemcpy(pathCounter_d, &pathCounter, sizeof(unsigned int), hipMemcpyHostToDevice);
        }
        
        int supportsCoopLaunch = 0;
        int dev = 0;
        hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev); 
        if( supportsCoopLaunch != 1)
            throw std::runtime_error("Cooperative Launch is not supported on this machine configuration.");

        hipEvent_t start, stop;
        hipDeviceSynchronize();
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        fbArgs.minimum=minimum_d;
        fbArgs.counters=counters_d;
        fbArgs.first_to_dequeue_global=first_to_dequeue_global_d;
        fbArgs.NODES_PER_SM=NODES_PER_SM_d;
        if(config.useGlobalMemory){
            fbArgs.global_memory=global_memory_d;
            sharedMemNeeded=0;
        }
        void *kernel_args[] = {&fbArgs};
        if (config.useGlobalMemory){
            if (config.version == HYBRID && config.instance==PVC){
                GlobalWorkListParameterized_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, global_memory_d, k_d, kFound_d, NODES_PER_SM_d);
            } else if(config.version == HYBRID && config.instance==MVC) {
                //GlobalWorkList_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, global_memory_d, NODES_PER_SM_d);
                hipLaunchCooperativeKernel((void *)GlobalWorkList_global_kernel, numBlocks, numThreadsPerBlock, (void **) (&kernel_args), sharedMemNeeded);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                LocalStacksParameterized_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, graph_d, global_memory_d, k_d, kFound_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                LocalStacks_global_kernel <<< numBlocks , numThreadsPerBlock >>> (stacks_d, graph_d, minimum_d, global_memory_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            }
        } else {
            if (config.version == HYBRID && config.instance==PVC){
                GlobalWorkListParameterized_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, k_d, kFound_d, NODES_PER_SM_d);
            } else if(config.version == HYBRID && config.instance==MVC) {
                //GlobalWorkList_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, minimum_d, workList_d, graph_d, counters_d, first_to_dequeue_global_d, NODES_PER_SM_d);
                hipLaunchCooperativeKernel((void *)GlobalWorkList_shared_kernel, numBlocks, numThreadsPerBlock, (void **) (&kernel_args), sharedMemNeeded);
            } else if(config.version == STACK_ONLY && config.instance==PVC){
                LocalStacksParameterized_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, graph_d, k_d, kFound_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            } else if(config.version == STACK_ONLY && config.instance==MVC) {
                LocalStacks_shared_kernel <<< numBlocks , numThreadsPerBlock, sharedMemNeeded >>> (stacks_d, graph_d, minimum_d, counters_d, pathCounter_d, NODES_PER_SM_d, config.startingDepth);
            }
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipError_t err = hipDeviceSynchronize();
        if(err != hipSuccess) {
            printf("GPU Error: %s\n", hipGetErrorString(err));
            exit(1);
        }

        // Copy back result
        if(config.instance == PVC){
            hipMemcpy(&kFound, kFound_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        } else {
            hipMemcpy(&minimum, minimum_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        }

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Elapsed time: %fms \n", milliseconds);

        printResults(config, RemoveMaxMinimum, RemoveEdgeMinimum, elapsed_seconds_max.count(), elapsed_seconds_edge.count(), minimum, milliseconds, numBlocks, 
            numBlocksPerSm, numThreadsPerSM, graph.vertexNum-1, graph.edgeNum, kFound);

        #if USE_COUNTERS
        printCountersInFile(config,counters_d,numBlocks);
        printNodesPerSM(config,NODES_PER_SM_d,numOfMultiProcessors);
        hipFree(NODES_PER_SM);
        #endif

        if(config.instance == PVC){
            hipFree(k_d);
        }
        graph.del();
        hipFree(minimum_d);
        hipFree(counters_d);
        cudaFreeGraph(fbArgs.graph);

        cudaFreeStacks(fbArgs.stacks);
        
        #if USE_GLOBAL_MEMORY
        hipFree(global_memory_d);
    
        #endif

        if(config.version == HYBRID){
            cudaFreeWorkList(fbArgs.workList);
            hipFree(first_to_dequeue_global_d);
        } else {
            hipFree(pathCounter_d);
        }

    }

    if(config.instance == PVC){
        if(kFound){
            printf("\nMinimum is less than or equal to K: %u\n\n",k);
        } else {
            printf("\nMinimum is greater than K: %u\n\n",k);
        }
    } else {
        printf("\nSize of minimum vertex cover: %u\n\n", minimum);
    }

    return 0;
}