/*
 * This source code is distributed under the terms defined  
 * in the file bfstdcsc_main.c of this source distribution.
 */
/* 
 *  Breadth first search (BFS) 
 *  Single precision (float data type) 
 *  TurboBFS_CSC_TD:bfsgputdcsc_sc.cu
 * 
 *  This program computes the GPU-based parallel 
 *  top-down BFS (scalar) for unweighted graphs represented 
 *  by sparse adjacency matrices in the CSC format, including
 *  the computation of the S array to store the depth at 
 *  which each vertex is discovered.  
 *
 */

#include <cstdlib>
#include <iostream>
#include <cassert>
#include <cmath>

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "mis_kernels.h"

#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

extern "C"{
                 #include "bfstdcsc.h"

}

/* 
 * Function to compute a gpu-based parallel maximal matching for 
 * unweighted graphs represented by sparse adjacency matrices in CSC format.
 *  
 */
//int  mm_gpu_csc (int *IC_h,int *CP_h,int *m_h,int *_m_d,int *req_h,int *c_h,int nz,int n,int repetition, int exec_protocol){
int mis_gpu(struct Graph * graph,struct MIS * mis, int exec_protocol){
  
  float t_mis;
  float t_mis_t = 0.0;
  int i,dimGrid;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int *L_h=mis->L_h;

  int *CP_d = graph->graph_device.CP_d;
  int *IC_d = graph->graph_device.IC_d;
  int *L_d = mis->mis_device.L_d;
  int *c = mis->mis_device.c;
  int n = graph->N;
  int nz = graph->nz;
  int repetition = graph->repet;

  dimGrid = (n + THREADS_PER_BLOCK)/THREADS_PER_BLOCK;
  thrust::device_ptr<int> L_vec=thrust::device_pointer_cast(L_d);
  int L_size, L_sum=0;
  for (i = 0; i<repetition; i++){
    *c = 1;
    checkCudaErrors(hipMemset(L_d,0,sizeof(*L_d)*graph->N));
    while(*c){
      *c = 0;
      hipEventRecord(start);
      set_L<<<dimGrid,THREADS_PER_BLOCK>>>(CP_d, IC_d, L_d, c, graph->N);
      hipEventRecord(stop);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&t_mis,start,stop);
      t_mis_t += t_mis;
    }
    L_size = thrust::count(L_vec, L_vec+graph->N, 1);
    L_sum += L_size;
  }
  if (graph->seq)
    checkCudaErrors(hipMemcpy(L_h,L_d, graph->N*sizeof(*L_h),hipMemcpyDeviceToHost));
  int print_t = 1;
  if (graph->p){
    printf("mis_gpu::total time = %lfms \n",t_mis_t);
    printf("mis_gpu::average size = %lf \n", (float)L_sum/(float)graph->repet);
    printf("mis_gpu::average time = %lfms \n",t_mis_t/graph->repet);
  }
  return 0;
}//end bfs_gpu_td_csc_sc
